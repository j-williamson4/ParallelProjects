#include "hip/hip_runtime.h"
/*
Fractal code for CS 4380 / CS 5351

Copyright (c) 2016, Texas State University. All rights reserved.

Redistribution in source or binary form, with or without modification,
is not permitted. Use in source and binary forms, with or without
modification, is only permitted for academic use in CS 4380 or CS 5351
at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
*/

#include <cstdlib>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "cs43805351.h"

static const int ThreadsPerBlock = 512;

static const double Delta = 0.005491;
static const double xMid = 0.745796;
static const double yMid = 0.105089;

static __global__
void FractalKernel(const int gpu_frames, const int width, unsigned char pic_d[])
{
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < gpu_frames * (width * width)) {
    const int col = idx % width;
    const int row = (idx / width) % width;
    const int frame = idx / (width * width);
   //todo: compute a single pixel here
    const double delta = Delta * pow(0.99, frame);
    const double xMin = xMid - delta;
    const double yMin = yMid - delta;
    const double dw = 2.0 * delta / width;
  //  for (int row = 0; row < width; row++) {
      const double cy = -yMin - row * dw;
  //  for (int col = 0; col < width; col++) {
      const double cx = -xMin - col * dw;
      double x = cx;
      double y = cy;
      int depth = 256;
      double x2, y2;
      do {
        x2 = x * x;
        y2 = y * y;
        y = 2 * x * y + cy;
        x = x2 - y2 + cx;
        depth--;
      } while ((depth > 0) && ((x2 + y2) < 5.0));
      pic_d[idx] = (unsigned char)depth;
  }
}

unsigned char* GPU_Init(const int size)
{
  unsigned char* pic_d;
  if (hipSuccess != hipMalloc((void **)&pic_d, size)) {fprintf(stderr, "could not allocate memory\n"); exit(-1);}
  return pic_d;
}

void GPU_Exec(const int gpu_frames, const int width, unsigned char pic_d[])
{
  // call the kernel (and do nothing else)
   int blockAmount = (width*width*gpu_frames + ThreadsPerBlock - 1) / ThreadsPerBlock;
  FractalKernel<<<blockAmount,ThreadsPerBlock>>>(gpu_frames, width, pic_d);
}

void GPU_Fini(const int size, unsigned char pic[], unsigned char pic_d[])
{
  // copy the pixel data to the CPU and deallocate the GPU array
  if (hipSuccess != hipMemcpy(pic, pic_d, size, hipMemcpyDeviceToHost)) {fprintf(stderr, "copying from device failed\n"); exit(-1);}
  hipFree(pic_d);
}

